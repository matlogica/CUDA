#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#ifdef __HIPCC__
// This code is compiled by the CUDA compiler
#ifdef __CUDA_ARCH__
// GPU-specific code: __CUDA_ARCH__ is defined, so we are compiling for the device

typedef double Real;
#else
// CPU-specific code: __CUDA_ARCH__ is not defined, so we are compiling for the host

#include <aadc/idouble.h>
#include <aadc/aadc.h>
typedef idouble Real;

#define HOST_ONLY_CODE
#endif
#endif

#include <iostream>

// Analytics callable from CPU and GPU
__host__ __device__ void analytics(Real a, Real b, Real &c) {
        c = a * std::exp(std::sin(b));
}

__global__ void CUDAKernel(Real *a, Real *b, Real *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        analytics(a[index], b[index], c[index]);
    }
}

int CALC() {
    int n = 256; // Size of the vector
    Real *a, *b, *c; // Pointers for host memory
    Real *d_a, *d_b, *d_c; // Pointers for device memory

    // Allocate host memory
    a = (Real*)malloc(n * sizeof(Real));
    b = (Real*)malloc(n * sizeof(Real));
    c = (Real*)malloc(n * sizeof(Real));

    // Initialize arrays with some values
    for(int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc(&d_a, n * sizeof(Real));
    hipMalloc(&d_b, n * sizeof(Real));
    hipMalloc(&d_c, n * sizeof(Real));

    // Copy inputs to device
    hipMemcpy(d_a, a, n * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(Real), hipMemcpyHostToDevice);

    // Launch kernel on default stream without arguments
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    CUDAKernel<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(c, d_c, n * sizeof(Real), hipMemcpyDeviceToHost);

    // Print results
    for(int i = 0; i < n; i++) {
        std::cout << a[i] << " " << b[i] << " " << c[i] << std::endl;
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    std::cout << "CUDA kernel executed successfully" << std::endl;

#ifdef HOST_ONLY_CODE
    std::cout << "Running AADC" << std::endl;

    /// AADC code
    
    auto da = (Real*)malloc(n * sizeof(Real));
    auto db = (Real*)malloc(n * sizeof(Real));


    typedef __m256d mmType;

    aadc::AADCFunctions<mmType> aadc_func;    

    aadc_func.startRecording();
    // Mark 1st elements of a and b as input
    aadc::AADCArgument arg_a(a[0].markAsInput()), arg_b(b[0].markAsInput());
    
    analytics(a[0], b[0], c[0]); // Call analytics function on 1st instance of data

    // Mark 1st element of c as output
    aadc::AADCResult res_c(c[0].markAsOutput());
    aadc_func.stopRecording();

    // Execute AADC kernel with derivatives

    int num_avx_elements = n / aadc::mmSize<mmType>();

    std::shared_ptr<aadc::AADCWorkSpace<mmType> > ws(aadc_func.createWorkSpace());

    for(int i = 0; i < num_avx_elements; i++) {
        for (int avx_i = 0; avx_i < aadc::mmSize<mmType>(); ++avx_i) {
            ws->valp(arg_a)[avx_i] = AAD_PASSIVE(a[i * aadc::mmSize<mmType>() + avx_i]);
            ws->valp(arg_b)[avx_i] = AAD_PASSIVE(b[i * aadc::mmSize<mmType>() + avx_i]);
        }
        aadc_func.forward(*ws);
        for (int avx_i = 0; avx_i < aadc::mmSize<mmType>(); ++avx_i) {
            c[i * aadc::mmSize<mmType>() + avx_i] = ws->valp(res_c)[avx_i];
        }
        ws->setDiff(res_c, 1.0);
        aadc_func.reverse(*ws);

        for (int avx_i = 0; avx_i < aadc::mmSize<mmType>(); ++avx_i) {
            da[i * aadc::mmSize<mmType>() + avx_i] = ws->diffp(arg_a)[avx_i];
            db[i * aadc::mmSize<mmType>() + avx_i] = ws->diffp(arg_b)[avx_i];
        }
    }

    // Print results
    for(int i = 0; i < n; i++) {
        std::cout << a[i] << " " << b[i] << " " << c[i] << " " << da[i] << " " << db[i] << std::endl;
    }

    // Free host memory
    free(da);
    free(db);

#endif

    // Free host memory
    free(a);
    free(b);
    free(c);


    return 0;
}
